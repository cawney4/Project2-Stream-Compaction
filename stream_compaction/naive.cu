#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"


namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        // Kernel that does a naive INCLUSIVE scan
        __global__ void kernNaiveScan(int n, int levelPower, int *odata, const int *idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (index >= n) {
                return;
            }

            if (index >= levelPower) {
                odata[index] = idata[index - levelPower] + idata[index];
            }
            else {
                odata[index] = idata[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

            // Initialize blockSize and fullBlocksPerGrid
            int blockSize = 128;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // Initialize variables and device arrays
            int totalLevels = ilog2ceil(n);
            int *dev_write;
            int *dev_read;

            // Allocate device arrays
            hipMalloc((void**) &dev_write, n * sizeof(int));
            checkCUDAError("hipMalloc dev_write failed!");

            hipMalloc((void**) &dev_read, n * sizeof(int));
            checkCUDAError("hipMalloc dev_read failed!");

            // Copy input data into dev_read
            hipMemcpy(dev_read, idata, sizeof(int) * n, hipMemcpyHostToDevice);


            timer().startGpuTimer();
            // TODO     
            
            // Go through the levels of Naive scan
            for (unsigned int level = 1; level <= totalLevels; level++) {
                int levelPower = pow(2, level - 1);

                // invoke kernel
                kernNaiveScan << <fullBlocksPerGrid, blockSize >> >(n, levelPower, dev_write, dev_read);

                // Ping-pong write and read arrays
                int* temp = dev_write;
                dev_write = dev_read;
                dev_read = temp;
            }

            // Copy final values into temporary array
            int* tempArray = new int[n];
            hipMemcpy(tempArray, dev_read, sizeof(int) * n, hipMemcpyDeviceToHost);

            // Copy values from tempArray while shifting values to convert inclusive scan to exclusive scan
            odata[0] = 0;
            for (int i = 1; i < n; i++) {
                odata[i] = tempArray[i - 1];
            }
            
            timer().endGpuTimer();

            // Free memory
            delete tempArray;
            hipFree(dev_write);
            hipFree(dev_read);

        }
    }
}
