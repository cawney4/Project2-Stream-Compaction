#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Kernel that does a up-sweep
        __global__ void kernUpSweep(int n, int levelPowerOne, int levelPower,  int *odata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;

            int divide = index / levelPowerOne; 
            if (index - (divide * levelPowerOne) == 0) {
                odata[index + levelPowerOne - 1] += odata[index + levelPower - 1];
            }

        }

        // Kernel that does a down-sweep
        __global__ void kernDownSweep(int n, int levelPowerPlusOne, int levelPower, int *odata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;

            int divide = index / levelPowerPlusOne;
            if (index - (divide * levelPowerPlusOne) == 0) {
                int temp = odata[index + levelPower - 1];
                odata[index + levelPower - 1] = odata[index + levelPowerPlusOne - 1];
                odata[index + levelPowerPlusOne - 1] += temp;
            }
        }

        /**
        * Performs prefix-sum (aka scan) on idata, storing the result into odata.
        */
        void scan(int n, int *odata, const int *idata) {
            scan(n, odata, idata, true);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, const bool time) {

            // Initialize blockSize and fullBlocksPerGrid
            int blockSize = 128;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            
            // Initialize variables and device arrays
            int totalLevels = ilog2ceil(n);
            int arraySize = pow(2, totalLevels); // To handle non-power of two lengths
            int *dev_array;

            // Allocate device array. 
            hipMalloc((void**) &dev_array, arraySize * sizeof(int));
            checkCUDAError("hipMalloc dev_array failed!");

            // Copy input data into dev_read
            hipMemcpy(dev_array, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            if (time) {
                timer().startGpuTimer();
            }
            // TODO
            // Go through the levels for Up Sweep
            for (unsigned int level = 0; level <= totalLevels; level++) {
                int levelPowerOne = pow(2, level + 1);
                int levelPower = pow(2, level);

                // invoke kernel
                kernUpSweep << <fullBlocksPerGrid, blockSize >> >(n, levelPowerOne, levelPower, dev_array);

            }

            // Copy values to a temporary array
            int* temp_array = new int[arraySize];
            hipMemcpy(temp_array, dev_array, sizeof(int) * arraySize, hipMemcpyDeviceToHost);

            // Set the last element to zero
            temp_array[arraySize - 1] = 0;

            // Copy array back to GPU
            hipMemcpy(dev_array, temp_array, sizeof(int) * arraySize, hipMemcpyHostToDevice);

            // Go through the levels for Down Sweep
            for (int level = totalLevels - 1; level >= 0; level--) {
                int levelPowerPlusOne = pow(2, level + 1);
                int levelPower = pow(2, level);

                // invoke kernel
                kernDownSweep << <fullBlocksPerGrid, blockSize >> >(n, levelPowerPlusOne, levelPower, dev_array);

            }

            // Copy data from GPU to output array
            hipMemcpy(odata, dev_array, sizeof(int) * n, hipMemcpyDeviceToHost);

            if (time) {
                timer().endGpuTimer();
            }

            // Free memory
            hipFree(dev_array);
            delete temp_array;
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int blockSize = 256;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // Device arrays
            int *dev_inData;
            int *dev_outData;
            int *dev_bool;
            int *dev_scan;

            // Allocate device array. 
            hipMalloc((void**) &dev_inData, n * sizeof(int));
            checkCUDAError("hipMalloc dev_inData failed!");

            hipMalloc((void**) &dev_outData, n * sizeof(int));
            checkCUDAError("hipMalloc dev_outData failed!");

            hipMalloc((void**) &dev_bool, n * sizeof(int));
            checkCUDAError("hipMalloc dev_bool failed!");

            hipMalloc((void**) &dev_scan, n * sizeof(int));
            checkCUDAError("hipMalloc dev_scan failed!");

            timer().startGpuTimer();
            // TODO
            
            // Map to booleans
            hipMemcpy(dev_inData, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            Common::kernMapToBoolean << < fullBlocksPerGrid, blockSize >> > (n, dev_bool, dev_inData);

            // Create host arrays that will be passed into scan
            int *scan_inData = new int[n];
            int *scan_outData = new int[n];
            hipMemcpy(scan_inData, dev_bool, sizeof(int) * n, hipMemcpyDeviceToHost);

            bool lastOne = scan_inData[n - 1]; // Remember if last bool is a 1. Will be used later.

            // Scan
            scan(n, scan_outData, scan_inData, false);

            // Use result from scan to find how many elements are compacted
            int count = scan_outData[n - 1];
            if (lastOne) {
                count++;
            }

            // Copy scan result to device
            hipMemcpy(dev_scan, scan_outData, sizeof(int) * n, hipMemcpyHostToDevice);

            // Perform scatter
            Common::kernScatter << < fullBlocksPerGrid, blockSize >> > (n, dev_outData,
                                                                        dev_inData, dev_bool, dev_scan);

            // Copy result to CPU
            hipMemcpy(odata, dev_outData, sizeof(int) * n, hipMemcpyDeviceToHost);

            timer().endGpuTimer();

            // Free memory
            hipFree(dev_inData);
            hipFree(dev_bool);
            hipFree(dev_scan);

            delete scan_inData;
            delete scan_outData;

            return count;
        }
    }
}
